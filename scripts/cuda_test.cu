
#include <hip/hip_runtime.h>
#include <iostream>
int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    std::cout << "CUDA devices: " << deviceCount << std::endl;
    return 0;
}
